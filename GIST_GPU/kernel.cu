#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <opencv2\opencv.hpp>
#include <iostream>
#include <hipfft/hipfft.h>

using namespace std;
using namespace cv;

# define PI           3.14159265358979323846
# define N			  256

hipError_t cudaStatus;

bool TIMER_FLG = 1;

const int numOfScales = 4;
const int numOfOrient = 8;
const int numOfFilters = numOfScales*numOfOrient;
const int filterSizeX = N;
const int filterSizeY = N;	
const int width = N;
const int height = N;	

float *dev_G_data = 0; 

hipfftHandle fft_plan = NULL, 
			ifft_plan = NULL;

__global__ void gaborCalcKernel(int numOfFilters, float* gaborParams, float* G)
{
    //Declaration
	int width = gridDim.x*blockDim.x;
	int height = gridDim.y*blockDim.y;
	int curX = blockIdx.x*blockDim.x+threadIdx.x;
	int curY = blockIdx.y*blockDim.y+threadIdx.y;
	int indx = curY*width+curX;
	int fx, fy;
	float fr1, t1, tr;

	//FFT Shift
	if (curX < width/2.0) fx = curX + width/2.0;
		else fx = curX - width/2.0;
	if (curY < height/2.0) fy = curY + height/2.0;
		else fy = curY - height/2.0;
	fx -= width/2.0;
	fy -= height/2.0;

	//Precalculate fr & t
	fr1 = sqrtf(fx*fx + fy*fy);
    t1 = atan2f(fy, fx);

	//Calculate Gabor Filter values
	for (int n=0; n<numOfFilters; n++)
	{

		tr = t1+gaborParams[n*4+3]; 
		if(tr < -PI) tr += 2.0f*PI;
		else if (tr > PI) tr -= 2.0f*PI;
			
		G[n*width*height+indx] = exp(-10.0f*gaborParams[n*4]*(fr1/height/gaborParams[n*4+1]-1)*
				(fr1/width/gaborParams[n*4+1]-1)-2.0f*gaborParams[n*4+2]*PI*tr*tr);
	}
}

__global__ void multiplyKernel(float* img_fft, float* G, float* res, int k)
{
	int width = gridDim.x*blockDim.x;
	int height = gridDim.y*blockDim.y;
	int curX = blockIdx.x*blockDim.x+threadIdx.x;
	int curY = blockIdx.y*blockDim.y+threadIdx.y;
	int indx = curY*width+curX;
	
	res[indx*2+0] = img_fft[indx*2+0]*G[k*width*height+indx];
	res[indx*2+1] = img_fft[indx*2+1]*G[k*width*height+indx];
}

float* generateGaborFiltersGPU()
{
    float *dev_gaborParams = 0;
	hipEvent_t start, stop;
	hipEvent_t full_start, full_stop;
	float elapsed_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&full_start);
	hipEventCreate(&full_stop);

	hipEventRecord(start);
	hipEventSynchronize(start);
	
	//Generate Gabor Filter parameters
	float gaborParams[numOfFilters][4]; 
	for (int i=0; i<numOfScales; i++)
		for (int j=0; j<numOfOrient; j++)
		{
			gaborParams[i*numOfOrient+j][0] = 0.35;
			gaborParams[i*numOfOrient+j][1] = 0.3/powf(1.85,i);
			gaborParams[i*numOfOrient+j][2] = 16.0*numOfOrient*numOfOrient/32.0/32.0;
			gaborParams[i*numOfOrient+j][3] = PI/numOfOrient*j;			
		}

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Gabor param time: " << elapsed_time << endl;

	hipEventRecord(start);
	hipEventSynchronize(start);
	//Gabor Calculations of CUDA
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)   
	cudaStatus = hipMalloc((void**)&dev_gaborParams, numOfFilters*4*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_G_data, numOfFilters*width*height*sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
	

    // Copy input vectors from host memory to GPU buffers.
	
	cudaStatus = hipMemcpy(dev_gaborParams, &gaborParams[0][0], numOfFilters*4*sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Memcpy time: " << elapsed_time << endl;
    // Launch a kernel on the GPU with one thread for each element.
	hipEventRecord(start);
	hipEventSynchronize(start);
	gaborCalcKernel <<< dim3(width/32, height/32), dim3(32, 32)>>> (numOfFilters, dev_gaborParams, dev_G_data);
	
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
   hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Gabor calc time: " << elapsed_time << endl;

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
			
    // Copy output vector from GPU buffer to host memory.

	float *G_data = (float *) malloc(numOfFilters*width*height*sizeof(float));
    cudaStatus = hipMemcpy(G_data, dev_G_data, numOfFilters*width*height*sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	
	// Copy G data to Mat vector
	/*for (int n=0; n<numOfFilters; n++)
	{
		Mat G0(width, height, CV_32F);
		float* G_ptr = (float*)G0.data;
		memcpy(G_ptr, &G_data[n*width*height], width*height*sizeof(float));
		G.push_back(G0);
	}*/

Error:
    hipFree(dev_gaborParams);
	//hipFree(dev_G_data);
	return G_data;
}



float* calcGistGPU(Mat img, float* G)
{
	//Declarations
	hipfftComplex *out_data, *dev_resComplex;
	float *dev_fft_img, *dev_res;
	hipfftReal *in_data;
	float* result = (float *) malloc(numOfFilters*width*height*sizeof(float));

	hipEvent_t start, stop;
	hipEvent_t full_start, full_stop;
	float elapsed_time = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&full_start);
	hipEventCreate(&full_stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

	//Resize and take pointer
	hipEventRecord(full_start);
	hipEventSynchronize(full_start);

	hipEventRecord(start);
	hipEventSynchronize(start);
	resize(img, img, Size(width, height));
	img.convertTo(img,CV_32F);
	img /= 256.0;
	float* img_ptr = (float*)img.data;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Resize time: " << elapsed_time << endl;

	//Data for FFTW
		
	// Allocate GPU buffers
	hipEventRecord(start);
	hipEventSynchronize(start);
	
	hipMalloc((void**)&in_data, width*height*sizeof(hipfftReal));
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMalloc failed!");

	hipMalloc((void**)&out_data, width*height*sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMalloc failed!");

	hipMalloc((void**)&dev_res, width*height*sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMalloc failed!");

	hipMalloc((void**)&dev_fft_img, 2*width*height*sizeof(float));
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMalloc failed!");

	hipMalloc((void**)&dev_resComplex, width*height*sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "hipMalloc failed!");
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Malloc time: " << elapsed_time << endl;
    

	//Copy data to GPU
	hipEventRecord(start);
	hipEventSynchronize(start);
	cudaStatus = hipMemcpy(in_data, img_ptr, width*height*sizeof(hipfftReal), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
        fprintf(stderr, "CPU->GPU hipMemcpy failed!");
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Memcpy time: " << elapsed_time << endl;
    

	//Create Plans FFT & IFFT
	hipEventRecord(start);
	hipEventSynchronize(start);
	if (fft_plan == NULL)
		hipfftPlan2d(&fft_plan, width, height, HIPFFT_R2C);
	if (ifft_plan == NULL)
		hipfftPlan2d(&ifft_plan, width, height, HIPFFT_C2R);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Create FFT plans time: " << elapsed_time << endl;

	//Execute Forward FFT
	hipEventRecord(start);
	hipEventSynchronize(start);
	hipfftExecR2C(fft_plan, in_data, out_data);
	hipDeviceSynchronize();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time, start, stop);
	if (TIMER_FLG) cout << "Calculate FFT time: " << elapsed_time << endl;

	//Copy Data to float pointer on GPU    
	cudaStatus = hipMemcpy(dev_fft_img, out_data, width*height*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) 
				fprintf(stderr, "GPU->CPU hipMemcpy failed!");
hipEventRecord(start);
			hipEventSynchronize(start);
		// Filter Image with Gabor filters in FFT Domain 
		for(int k = 0; k < 32; k++)
	    {	

			multiplyKernel <<< dim3(width/32, height/32), dim3(32, 32)>>> (dev_fft_img, dev_G_data, dev_res, k);
			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
				fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess)
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);

			//Copy data to Complex ppointer on GPU
			cudaStatus = hipMemcpy(dev_resComplex, dev_res, width*height*sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
			if (cudaStatus != hipSuccess) 
				fprintf(stderr, "GPU->GPU hipMemcpy failed!");
						
	       //Execute Forward FFT
			hipfftExecC2R(ifft_plan, dev_resComplex, in_data);
			hipDeviceSynchronize();

			//Copy data from GPU	
			cudaStatus = hipMemcpy(&result[k*width*height], in_data, width*height*sizeof(hipfftReal), hipMemcpyDeviceToHost);
			if (cudaStatus != hipSuccess)
				fprintf(stderr, "GPU->CPU hipMemcpy failed!\n");

			

	    }
		hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsed_time, start, stop);
			if (TIMER_FLG) cout << "FFT Convolution time: " << elapsed_time << endl;
		//Clean trash
		hipEventRecord(start);
		hipEventSynchronize(start);
		//hipfftDestroy(fft_plan);
		//hipfftDestroy(ifft_plan);
		hipFree(in_data);
		hipFree(out_data);
		//free(buffer);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		if (TIMER_FLG) cout << "Clean time: " << elapsed_time << endl;

		hipEventRecord(full_stop);
		hipEventSynchronize(full_stop);
		hipEventElapsedTime(&elapsed_time, full_start, full_stop);
		//cout << "Full calcGist time: " << elapsed_time << endl;

	return result;
}